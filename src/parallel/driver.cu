#include "hip/hip_runtime.h"
#include "driver.h"
#include <iostream>
#include <stdio.h>
namespace mincut {
  __global__ void reduce(float *d_slinks, float *d_capacity, float *d_flow, int i, int xsize, int ysize) {
    // Find paths of length i and remove bottleneck
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    int xboundmin = blockDim.x * blockIdx.x;
    int xboundmax = blockDim.x * (blockIdx.x + 1) - 1;
    int yboundmin = blockDim.y * blockIdx.y;
    int yboundmax = blockDim.y * (blockIdx.y + 1) - 1;

    int index = x * ysize + y;
    if (i == 1) {
      int s_weight = d_slinks[index];
      int t_weight = d_capacity[5*index + 4];
      if (s_weight > t_weight) {
	d_capacity[5*index + 4] = 0;
	d_slinks[index] -= t_weight;
      } else {
	d_capacity[5*index + 4] -= s_weight;
	d_slinks[index] = 0;
      }
    }
    if (i == 2) {
      float weights[4];
      int next_pos[2];
      int tempindex = index;
      weights[0] = d_slinks[index];
      char offsets[][8] = { {D,L}, {D,R}, {U,L}, {U,R}, {R, U}, {R, D}, {L, D}, {L, U} };
      for (int j = 0; j < 8; j++) {
	char mincappos = 0;
	for (int k = 1; k < 3; k++) {
	  if (offsets[j][k-1] == D) {
	    next_pos[k-1] = index + ysize;
	  } else if (offsets[j][k-1] == L) {
	    next_pos[k-1] = index - xsize;
	  } else if (offsets[j][k-1] == R) {
	    next_pos[k-1] = index + xsize;
	  } else if (offsets[j][k-1] == U) {
	    next_pos[k-1] = index - ysize;
	  }
	  weights[k] = d_capacity[5*index + offsets[j][k-1]] - d_flow[5*index + offsets[j][k-1]];
	  if (weights[k] < 0) {
	    weights[k] = 0;
	  }
	  if (weights[k] < weights[mincappos]) {
	    mincappos = k;
	  }
	  index = next_pos[k-1];
	}
	weights[3] = d_capacity[5*index + 4] - d_flow[5*index + 4];
	bool good = true;
	
	for (int i = 0; i < 2; i++){
	  
	}
      }
    }

    return;
  }

  float mincut (Graph *g) {

    float *d_slinks, *d_capacity, *d_flow;

    hipMalloc((void **) &d_capacity,5*IMAGE_SIZE*FLOAT_SIZE);
    hipMalloc((void **) &d_flow,5*IMAGE_SIZE *FLOAT_SIZE);
    hipMalloc((void **) &d_slinks,IMAGE_SIZE * FLOAT_SIZE);

    hipMemcpy(d_capacity,g->m_capacity,5*IMAGE_SIZE*FLOAT_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_flow,g->m_flow,5*IMAGE_SIZE*FLOAT_SIZE,hipMemcpyHostToDevice);
    hipMemcpy(d_slinks,g->m_slinks,IMAGE_SIZE * FLOAT_SIZE,hipMemcpyHostToDevice);

    hipEvent_t seq_start_event, seq_stop_event;
    hipEventCreate(&seq_start_event);
    hipEventCreate(&seq_stop_event);
    hipEventRecord(seq_start_event, 0);


    for (int i = 0; i < 2; i++) {
      reduce<<<dim3(20,20),dim3(10,10)>>> (d_capacity, d_flow, d_slinks, i, g->x, g->y);
    }

    hipEventRecord(seq_stop_event, 0);
    hipEventSynchronize(seq_stop_event);

    hipDeviceSynchronize();
    float seq_time= 0;
    hipEventElapsedTime(&seq_time, seq_start_event, seq_stop_event);
    std::cout << "Time taken for reduce :" << seq_time << std::endl;
    return 0.0f;

  }


}
